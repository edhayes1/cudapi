
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <chrono>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
using namespace std;

__device__ unsigned int reduce_sum(unsigned int in)
{
    extern __shared__ unsigned int sdata[];

    // Perform first level of reduction:
    // - Write to shared memory
    unsigned int ltid = threadIdx.x;

    sdata[ltid] = in;
    __syncthreads();

    // Do reduction in shared mem
    for (unsigned int s = blockDim.x / 2 ; s > 0 ; s >>= 1)
    {
        if (ltid < s)
        {
            sdata[ltid] += sdata[ltid + s];
        }

        __syncthreads();
    }

    return sdata[0];
}

__global__ void mykernel(int vectorsize, int *count, double *rands) 
{
	int id = blockIdx.x *blockDim.x + threadIdx.x;

	int step = gridDim.x * blockDim.x;

	const double *rand1 = rands + id;
    const double *rand2 = rand1 + vectorsize;

    int tempcount = 0;

	for (int i = 0; i < vectorsize; i += step, rand1 +=step, rand2 += step)
	{
		double x = *rand1;
		double y = *rand2;
		if(((x*x)+(y*y)) < 1 )
			tempcount++;
	}
	tempcount = reduce_sum(tempcount);

	if (threadIdx.x == 0)
    {
        count[blockIdx.x] = tempcount;
    }
}


double * createrands(double vectorsize)
{
	hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_MTGP32);
    double *rands = 0;
    hipMalloc((void **)&rands, 2* vectorsize * sizeof(double));
    hiprandSetPseudoRandomGeneratorSeed(prng, 1337);
    hiprandGenerateUniformDouble(prng, (double *)rands, 2 * vectorsize);
    hiprandDestroyGenerator(prng);

    return rands;
}

int main(void) 
{
	auto t_start = std::chrono::high_resolution_clock::now();
	//int numgpus = 2;
	double vectorsize = 33553920;
	// cin >> vectorsize;
	int blocksize = 1024;

	int gridsize = ceil(vectorsize/blocksize);
	size_t sharedmemsize = blocksize * sizeof(int);

	int *count1, *count2, *cuda_count1, *cuda_count2;	
	
	count1 = (int *)malloc (gridsize * sizeof(int));
	count2 = (int *)malloc (gridsize * sizeof(int));

	//1st gpu
	hipSetDevice(0);
	double *rands1 = createrands(vectorsize);
	hipMalloc((void **)&cuda_count1, gridsize *sizeof(int));

	mykernel <<<gridsize, blocksize, sharedmemsize>>>(vectorsize, cuda_count1, rands1);
	//2nd gpu
	hipSetDevice(1);
	double *rands2 = createrands(vectorsize);
	hipMalloc((void **)&cuda_count2, gridsize *sizeof(int));

	mykernel <<<gridsize, blocksize, sharedmemsize>>>(vectorsize, cuda_count2, rands2);

	//1st gpu
	hipSetDevice(0);
	if (hipMemcpy (count1, cuda_count1, gridsize *sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
		printf("failed to cpy back 1\n");
	hipFree(cuda_count1);

	//2nd gpu
	hipSetDevice(1);
	if (hipMemcpy (count2, cuda_count2, gridsize *sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
		printf("failed to cpy back 2\n");
	hipFree(cuda_count2);

	int totalcount = 0;
	for (int i = 0; i < gridsize; i ++)
	{
		totalcount += count1[i];
	}
	for (int i = 0; i < gridsize; i ++)
	{
		totalcount += count2[i];
	}

	printf("count = %d\n", totalcount);
	float ratio = totalcount / (2*vectorsize);
	printf("pi =  %.15f \n", (ratio * 4));

	auto t_end = std::chrono::high_resolution_clock::now();

    printf("duration: %f\n", (std::chrono::duration<double, std::milli>(t_end-t_start).count()/1000));

	return 0;
}
